
#include "hip/hip_runtime.h"
#include ""
#include "book.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define SIZE (10*1024*1024)

float cuda_malloc_test(int size, bool up) //bool: �� �Ǵ� ������ ��Ÿ���� �ڷ���, �⺻���� false�� ��Ÿ��
{
	hipEvent_t start, stop;
	int* a, * dev_a;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	a = (int*)malloc(size * sizeof(*a));
	//HANDLE_NULL(a);
	hipMalloc((void**)&dev_a, size * sizeof(*dev_a));
	hipEventRecord(start, 0);
	for (int i = 0; i < 100; i++)
	{
		if (up)
		{
			hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
		}
		else
		{
			hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
		}
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	free(a);
	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}

float cuda_host_alloc_test(int size, bool up)
{
	hipEvent_t start, stop;
	int* a, * dev_a;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipHostAlloc((void**)&a, size * sizeof(*a), hipHostMallocDefault);

	hipMalloc((void**)&dev_a, size * sizeof(*dev_a));

	hipEventRecord(start, 0);
	for (int i = 0; i < 100; i++)
	{
		if (up)
		{
			hipMemcpy(dev_a, a, size * sizeof(*a), hipMemcpyHostToDevice);
		}
		else
		{
			hipMemcpy(a, dev_a, size * sizeof(*a), hipMemcpyDeviceToHost);
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipHostFree(a);
	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}

int main()
{
	float elapsedTime;
	float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;

	elapsedTime = cuda_malloc_test(SIZE, true);
	printf("Time using hipMalloc: %3.lf ms\n", elapsedTime);
	printf("\tMB/s during copy up: %3.lf\n", MB / (elapsedTime / 1000));

	elapsedTime = cuda_malloc_test(SIZE, false);
	printf("Time using hipMalloc: %3.lf ms\n", elapsedTime);
	printf("\tMB/s during copy down: %3.lf\n", MB / (elapsedTime / 1000));

	elapsedTime = cuda_host_alloc_test(SIZE, true);
	printf("Time using hipHostAlloc: %3.lf ms\n", elapsedTime);
	printf("\tMB/s during copy up: %3.lf ms\n", MB / (elapsedTime / 1000));

	elapsedTime = cuda_host_alloc_test(SIZE, false);
	printf("Time using hipHostAlloc: %3.lf ms\n", elapsedTime);
	printf("\tMB/s during copy down: %3.lf ms\n", MB / (elapsedTime / 1000));
}

