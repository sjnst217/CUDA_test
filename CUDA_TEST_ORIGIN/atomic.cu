#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "book.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define SIZE (100 * 1024 * 1024)

__global__ void histo_kernel(unsigned char* buffer, long size, unsigned int* histo)
{
	__shared__ unsigned int temp[256];
	temp[threadIdx.x] = 0;
	__syncthreads();

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	while (i < size)
	{
		atomicAdd(&temp[buffer[i]], 1);
		i += offset;
	}

	__syncthreads();
	atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}

int main()
{
	unsigned char* buffer = (unsigned char*)big_random_block(SIZE); // 100MB�� �������� �����͸� ����

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	unsigned char* dev_buffer;
	unsigned int* dev_histo;
	hipMalloc((void**)&dev_buffer, SIZE);
	hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_histo, 256 * sizeof(long));
	hipMemset(dev_histo, 0, 256 * sizeof(int));

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int blocks = prop.multiProcessorCount;
	histo_kernel << <blocks * 2, 256 >> > (dev_buffer, SIZE, dev_histo);

	unsigned int histo[256]; 

	hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to generate: %3.lf ms\n", elapsedTime);


	long histoCount = 0;
	for (int i = 0; i < 256; i++)
	{
		histoCount += histo[i];
	}
	printf("Histogram Sum: %ld\n", histoCount);

	for (int i = 0; i < SIZE; i++)
	{
		histo[buffer[i]]--;
	}
	for (int i = 0; i < 256; i++)
	{
		if (histo[i] != 0)
		{
			printf("Failure at %d!\n", i);
		}
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(dev_histo);
	hipFree(dev_buffer);
	free(buffer);
	return 0;
}
//�̿�