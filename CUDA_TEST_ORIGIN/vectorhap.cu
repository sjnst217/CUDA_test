
#include "hip/hip_runtime.h"


#include <stdio.h>

#define arraySize  (33 * 1024) //->33792


void addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b) //GPU에서 실행되는 코드
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x; //임의의 긴 벡터의 GPU 합(thread 기준 전체 수)
    while(tid < arraySize) // arraySize 만큼의 계산을 하기 위해서 반복해줌 
    {
        c[tid] = a[tid] + b[tid]; //일반적인 벡터합 과정
        tid += blockDim.x * gridDim.x; //벡터합 과정중 arraySize보다 큰 tid는 수행하지 못하도록 while문을 종료시키기 위함
    }
    // 아래 코드는 오류
    /*if (tid < arraySize)
    {
        c[tid] = a[tid] + b[tid];
    }*/
    // 오류 이유 -> (GPU가 한번에 계산 가능한 16384만큼의 tid까지만 계산하고 그 이후의 33729까지의 계산에서f c[tid]를 계산하지 않는 코드 이므로)
}

void test1()
{
    int a[arraySize] = { 0 };
    int b[arraySize] = { 0 };
    int c[arraySize] = { 0 };

    for (int i = 0; i < arraySize; i++)
    {
        a[i] = i;
        b[i] = i + i;
    }
    // Add vectors in parallel.
    addWithCuda(c, a, b, arraySize);
    printf("\n");
    
    bool success = true;
    for (int i = 0; i < arraySize; i++)
    {
        if ((a[i] + b[i]) != c[i])
        {
            printf("Error: %d + %d != %d\n", a[i], b[i], c[i]);
            success = false;
        }
    }
    if(success)
    {
        printf("We did it!\n");
    }
    
    for (int i = 16383; i < 16390; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    } //계산이 어디까지 되었는지 확인하는 코드

   
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipDeviceReset();
}

void test2()
{

}

int main()
{
    test1();

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;


    // Choose which GPU to run on, change this on a multi-GPU system.

    // Allocate GPU buffers for three vectors (two input, one output)    .
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    //       (block, thread)   (input or output) 
    addKernel <<<128, 128>>>(dev_c, dev_a, dev_b); //여기에서부터 GPU시작, 한번에 처리 가능한 수는 16384

    // Check for any errors launching the kernel

    //cudaDeviceSynchronize();
    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);



    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

}
