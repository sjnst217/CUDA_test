
#include "hip/hip_runtime.h"


#include <stdio.h>

void addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b) //GPU���� ����Ǵ� �ڵ�
{
    int i = threadIdx.x;
    int block = blockIdx.x;
    c[i] = a[i] + b[block];
}



int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    addWithCuda(c, a, b, arraySize); //�� ������ CPU�ε�
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipDeviceReset();
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;


    // Choose which GPU to run on, change this on a multi-GPU system.

    // Allocate GPU buffers for three vectors (two input, one output)    .
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    //       (block, thread)   (input or output) 
    addKernel <<<1, size >>> (dev_c, dev_a, dev_b); //���⿡������ GPU����

    // Check for any errors launching the kernel

    //cudaDeviceSynchronize();
    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

}
